#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib> // For std::atoi
#include <chrono> // For benchmarking
#include <hip/hip_runtime.h>
#include "../src/shared_memory_access.hpp"

using SharedMemoryAccess::Fields::c; // concentration
using SharedMemoryAccess::Fields::dt; // time step
using SharedMemoryAccess::Fields::timestep; // simulation time

using ArrayType = std::remove_reference_t<decltype(c)>; // type of array, like float[800][600]
constexpr std::size_t Rows = std::extent<ArrayType, 0>::value;
constexpr std::size_t Cols = std::extent<ArrayType, 1>::value;

ArrayType temp{0}; // local temporary array

__device__ __constant__ float d_dt;
__device__ __constant__ float d_timestep;

// Apply boundary conditions
__global__ void apply_boundary_conditions(float* d_c, size_t pitch) {
    constexpr float source_value = 1.0f;
    constexpr float sink_value = 0.0f;

    int j = threadIdx.x + blockIdx.x * blockDim.x;
    if (j < Cols) {
        float* row_start = (float*)((char*)d_c + 0 * pitch);
        row_start[j] = source_value;

        row_start = (float*)((char*)d_c + (Rows - 1) * pitch);
        row_start[j] = sink_value;
    }

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < Rows) {
        float* row_start = (float*)((char*)d_c + i * pitch);
        row_start[0] = row_start[1];
        row_start[Cols - 1] = row_start[Cols - 2];
    }
}

// Perform diffusion
__global__ void perform_diffusion(const float* d_c, float* d_temp, size_t pitch) {
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    if (i > 0 && i < Rows - 1 && j > 0 && j < Cols - 1) {
        float* current_row = (float*)((char*)d_c + i * pitch);
        float* upper_row = (float*)((char*)d_c + (i - 1) * pitch);
        float* lower_row = (float*)((char*)d_c + (i + 1) * pitch);

        float* temp_row = (float*)((char*)d_temp + i * pitch);
        temp_row[j] = current_row[j] + d_dt * (
            upper_row[j] + lower_row[j] +
            current_row[j - 1] + current_row[j + 1] -
            4 * current_row[j]
        );
    }
}

int main(int argc, char* argv[]) {
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <number of iterations> <update_every>" << std::endl;
        return 1;
    }
    int iterations = std::atoi(argv[1]);
    int update_every = std::atoi(argv[2]);
    if (iterations <= 0 || update_every <= 0) {
        std::cerr << "Number of iterations and update_every must be positive integers." << std::endl;
        return 1;
    }

    dim3 blockSize(16, 16);
    dim3 gridSize((Cols + blockSize.x - 1) / blockSize.x, (Rows + blockSize.y - 1) / blockSize.y);

    float* d_c;
    float* d_temp;
    size_t pitch; //width of a row in bytes when allocating 2D memory on a CUDA device

    {
    if (hipMallocPitch((void**)&d_c, &pitch, Cols * sizeof(float), Rows) != hipSuccess) {
        std::cerr << "Error: Failed to allocate device memory for d_c." << std::endl;
        return 1;
    }

    if (hipMallocPitch((void**)&d_temp, &pitch, Cols * sizeof(float), Rows) != hipSuccess) {
        std::cerr << "Error: Failed to allocate device memory for d_temp." << std::endl;
        hipFree(d_c);
        return 1;
    }

    if (hipMemcpyToSymbol(HIP_SYMBOL(d_dt), &dt, sizeof(float)) != hipSuccess) {
        std::cerr << "Error: Failed to copy dt to device constant memory." << std::endl;
        hipFree(d_c);
        hipFree(d_temp);
        return 1;
    }

    if (hipMemcpyToSymbol(HIP_SYMBOL(d_timestep), &timestep, sizeof(float)) != hipSuccess) {
        std::cerr << "Error: Failed to copy timestep to device constant memory." << std::endl;
        hipFree(d_c);
        hipFree(d_temp);
        return 1;
    }

    if (hipMemcpy2D(d_c, pitch, c, Cols * sizeof(float), Cols * sizeof(float), Rows, hipMemcpyHostToDevice) != hipSuccess) {
        std::cerr << "Error: Failed to copy c to device memory." << std::endl;
        hipFree(d_c);
        hipFree(d_temp);
        return 1;
    }
    }

    auto start_time = std::chrono::high_resolution_clock::now();

    for (int iter = 0; iter < iterations; ++iter) {
        
        perform_diffusion<<<gridSize, blockSize>>>(d_c, d_temp, pitch);
        // if (hipDeviceSynchronize() != hipSuccess) {
        //     std::cerr << "Error: Failed to synchronize after perform_diffusion kernel." << std::endl;
        //     hipFree(d_c);
        //     hipFree(d_temp);
        //     return 1;
        // }

        std::swap(d_c, d_temp);

        apply_boundary_conditions<<<(std::max(Rows, Cols) + 255) / 256, 256>>>(d_c, pitch);
        // if (hipDeviceSynchronize() != hipSuccess) {
        //     std::cerr << "Error: Failed to synchronize after apply_boundary_conditions kernel." << std::endl;
        //     hipFree(d_c);
        //     hipFree(d_temp);
        //     return 1;
        // }

        timestep += dt;
        // if (hipMemcpyToSymbol(HIP_SYMBOL(d_timestep), &timestep, sizeof(float)) != hipSuccess) {
        //     std::cerr << "Error: Failed to update timestep in device constant memory." << std::endl;
        //     hipFree(d_c);
        //     hipFree(d_temp);
        //     return 1;
        // }

        if ((iter + 1) % update_every == 0) {
            if (hipMemcpy2D(c, Cols * sizeof(float), d_c, pitch, Cols * sizeof(float), Rows, hipMemcpyDeviceToHost) != hipSuccess) {
                std::cerr << "Error: Failed to copy c from device to host." << std::endl;
                hipFree(d_c);
                hipFree(d_temp);
                return 1;
            }
        }
    }

    if (hipMemcpy2D(c, Cols * sizeof(float), d_c, pitch, Cols * sizeof(float), Rows, hipMemcpyDeviceToHost) != hipSuccess) {
        std::cerr << "Error: Failed to copy c from device to host." << std::endl;
        hipFree(d_c);
        hipFree(d_temp);
        return 1;
    }

    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_seconds = end_time - start_time;

    std::cout << "Done, " << iterations << " iterations in "
              << elapsed_seconds.count() << " seconds." << std::endl;

    hipFree(d_c);
    hipFree(d_temp);

    return 0;
}
