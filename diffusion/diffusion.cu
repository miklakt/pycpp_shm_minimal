#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib> // For std::atoi
#include <chrono> // For benchmarking
#include <hip/hip_runtime.h>
#include <memory> // For std::unique_ptr
#include "../src/shared_memory_access.hpp"

using SharedMemoryAccess::Fields::c; // concentration
using SharedMemoryAccess::Fields::dt; // time step
using SharedMemoryAccess::Fields::timestep; // simulation time

using ArrayType = std::remove_reference_t<decltype(c)>; // type of array, like float[800][600]
constexpr std::size_t Rows = std::extent<ArrayType, 0>::value;
constexpr std::size_t Cols = std::extent<ArrayType, 1>::value;

ArrayType temp{0}; // local temporary array

__device__ __constant__ float d_dt;
__device__ __constant__ float d_timestep;

// Pitch size width in bytes of the allocated memory for a single row, for memory alignment
// Must be the same for all arrays
size_t pitch;

// Define CUDA block and grid sizes
// For 2D stencil operations
dim3 blockSize(16, 16);
dim3 gridSize((Cols + blockSize.x - 1) / blockSize.x, (Rows + blockSize.y - 1) / blockSize.y);

// Define CUDA block and grid sizes
// For 1D linear operations
const int blockSize1D(256);
const int gridSize1D((std::max(Rows, Cols) + blockSize1D-1) / blockSize1D);

// Allocate device memory with automatic cleanup using std::unique_ptr
auto make_unique_ptr_cuda(){
    return std::unique_ptr<float, decltype(&hipFree)>(
        [&]{
            float* ptr; 
            hipMallocPitch((void**)&ptr, &pitch, Cols * sizeof(float), Rows); 
            return ptr;}(), 
            hipFree
        );
}

// Apply boundary conditions to grid edges
__global__ void apply_boundary_conditions(float* d_c, size_t pitch) {
    constexpr float source_value = 1.0f;
    constexpr float sink_value = 0.0f;

    int j = threadIdx.x + blockIdx.x * blockDim.x;
    if (j < Cols) {
        float* row_start = (float*)((char*)d_c + 0 * pitch);
        row_start[j] = source_value;

        row_start = (float*)((char*)d_c + (Rows - 1) * pitch);
        row_start[j] = sink_value;
    }

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < Rows) {
        float* row_start = (float*)((char*)d_c + i * pitch);
        row_start[0] = row_start[1];
        row_start[Cols - 1] = row_start[Cols - 2];
    }
}

// Perform diffusion step on the grid
__global__ void perform_diffusion(const float* d_c, float* d_temp, size_t pitch) {
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    if (i > 0 && i < Rows - 1 && j > 0 && j < Cols - 1) {
        float* current_row = (float*)((char*)d_c + i * pitch);
        float* upper_row = (float*)((char*)d_c + (i - 1) * pitch);
        float* lower_row = (float*)((char*)d_c + (i + 1) * pitch);

        float* temp_row = (float*)((char*)d_temp + i * pitch);
        temp_row[j] = current_row[j] + d_dt * (
            upper_row[j] + lower_row[j] +
            current_row[j - 1] + current_row[j + 1] -
            4 * current_row[j]
        );
    }
}


int main(int argc, char* argv[]) {
    // Validate and parse command-line arguments
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <number of iterations> <update_every>" << std::endl;
        return 1;
    }
    int iterations = std::atoi(argv[1]);
    int update_every = std::atoi(argv[2]);
    if (iterations <= 0 || update_every <= 0) {
        std::cerr << "Number of iterations and update_every must be positive integers." << std::endl;
        return 1;
    }

    // Allocate device memory
    auto d_c = make_unique_ptr_cuda();
    auto d_temp = make_unique_ptr_cuda();

    // Copy constants to device
    hipMemcpyToSymbol(HIP_SYMBOL(d_dt), &dt, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_timestep), &timestep, sizeof(float));

    // Copy initial data to device memory
    hipMemcpy2D(d_c.get(), pitch, c, Cols * sizeof(float), Cols * sizeof(float), Rows, hipMemcpyHostToDevice);

    // Start benchmarking
    auto start_time = std::chrono::high_resolution_clock::now();

    // Main computation loop
    for (int iter = 0; iter < iterations; ++iter) {
        perform_diffusion<<<gridSize, blockSize>>>(d_c.get(), d_temp.get(), pitch);
        apply_boundary_conditions<<<gridSize1D, blockSize1D>>>(d_temp.get(), pitch);

        std::swap(d_c, d_temp);
        timestep += dt;

        // Periodically update results on host
        if ((iter + 1) % update_every == 0) {
            hipDeviceSynchronize();
            hipMemcpyToSymbol(HIP_SYMBOL(d_timestep), &timestep, sizeof(float));
            hipMemcpy2D(c, Cols * sizeof(float), d_c.get(), pitch, Cols * sizeof(float), Rows, hipMemcpyDeviceToHost);
        }
    }

    hipDeviceSynchronize();
    hipMemcpyToSymbol(HIP_SYMBOL(d_timestep), &timestep, sizeof(float));
    hipMemcpy2D(c, Cols * sizeof(float), d_c.get(), pitch, Cols * sizeof(float), Rows, hipMemcpyDeviceToHost);

     // End benchmarking
    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_seconds = end_time - start_time;

    std::cout << "Done, " << iterations << " iterations in "
              << elapsed_seconds.count() << " seconds." << std::endl;

    return 0;
}
