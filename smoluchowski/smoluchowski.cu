#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdlib> // For std::atoi
#include <chrono>  // For benchmarking
#include <hip/hip_runtime.h>
#include <memory>  // For std::unique_ptr
#include "../src/shared_memory_access.hpp"

// Exposing Shared Memory fields
using SharedMemoryAccess::Fields::c;
using SharedMemoryAccess::Fields::c_next;
using SharedMemoryAccess::Fields::D_x;
using SharedMemoryAccess::Fields::D_y;
using SharedMemoryAccess::Fields::dU_x;
using SharedMemoryAccess::Fields::dU_y;
using SharedMemoryAccess::Fields::alpha_x;
using SharedMemoryAccess::Fields::alpha_y;
using SharedMemoryAccess::Fields::lambda_n;
using SharedMemoryAccess::Fields::lambda_s;
using SharedMemoryAccess::Fields::div_J;
using SharedMemoryAccess::Fields::dt;
using SharedMemoryAccess::Fields::timestep;

using ArrayType = std::remove_reference_t<decltype(c)>; // Type of array, like float[800][600]
constexpr std::size_t Rows = std::extent<ArrayType, 0>::value;
constexpr std::size_t Cols = std::extent<ArrayType, 1>::value;

__device__ __constant__ float d_dt;
__device__ __constant__ float d_timestep;

// Pitch size width in bytes of the allocated memory for a single row, for memory alignment
// Must be the same for all arrays
size_t pitch;

// Define CUDA block and grid sizes
// For 2D stencil operations
dim3 blockSize(16, 16);
dim3 gridSize((Cols + blockSize.x - 1) / blockSize.x, (Rows + blockSize.y - 1) / blockSize.y);

// Define CUDA block and grid sizes
// For 1D linear operations
const int blockSize1D(256);
const int gridSize1D((std::max(Rows, Cols) + blockSize1D-1) / blockSize1D);

// Allocate device memory with automatic cleanup using std::unique_ptr
auto make_unique_ptr_cuda(){
    return std::unique_ptr<float, decltype(&hipFree)>(
        [&]{
            float* ptr; 
            hipMallocPitch((void**)&ptr, &pitch, Cols * sizeof(float), Rows); 
            return ptr;}(), 
            hipFree
        );
}

auto make_unique_ptr_1D_column_vector_cuda(){
        return std::unique_ptr<float, decltype(&hipFree)>(
        [&]{
            float* ptr; 
            hipMalloc((void**)&ptr, Rows * sizeof(float)); 
            return ptr;}(), 
            hipFree
        );
}

// Apply boundary conditions to grid edges
__global__ void apply_boundary_conditions(float* d_c, size_t pitch) {
    constexpr float source_value = 1.0f;
    constexpr float sink_value = 0.0f;

    int j = threadIdx.x + blockIdx.x * blockDim.x;
    if (j < Cols) {
        float* row_start = (float*)((char*)d_c + 0 * pitch);
        row_start[j] = source_value;

        row_start = (float*)((char*)d_c + (Rows - 1) * pitch);
        row_start[j] = sink_value;
    }

    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < Rows) {
        float* row_start = (float*)((char*)d_c + i * pitch);
        row_start[0] = row_start[1];
        row_start[Cols - 1] = row_start[Cols - 2];
    }
}

#define ACCESS_2D(ptr, i, j, pitch) ((float*)((char*)ptr + (i) * pitch))[j]
// Drift-diffusion kernel
__global__ void drift_diffusion(
    float* d_c, float* d_c_next, 
    float* d_D_x, float* d_D_y, 
    float* d_alpha_x, float* d_alpha_y, 
    float* d_dU_x, float* d_dU_y, 
    float* d_lambda_n, float* d_lambda_s, 
    float* d_div_J, 
    size_t pitch) {
        
    int i = threadIdx.y + blockIdx.y * blockDim.y;
    int j = threadIdx.x + blockIdx.x * blockDim.x;

    if (i > 0 && i < Rows - 1 && j > 0 && j < Cols - 1) {
        // Extract neighboring concentrations
        float c_P = ACCESS_2D(d_c, i, j, pitch);
        float c_E = ACCESS_2D(d_c, i + 1, j, pitch);
        float c_W = ACCESS_2D(d_c, i - 1, j, pitch);
        float c_N = ACCESS_2D(d_c, i, j + 1, pitch);
        float c_S = ACCESS_2D(d_c, i, j - 1, pitch);

        // Concentration gradients
        float grad_c_e = c_E - c_P;
        float grad_c_w = c_P - c_W;
        float grad_c_n = c_N - c_P;
        float grad_c_s = c_P - c_S;

        // Diffusion fluxes
        float J_dif_e = -ACCESS_2D(d_D_x, i, j, pitch) * grad_c_e;
        float J_dif_w = -ACCESS_2D(d_D_x, i - 1, j, pitch) * grad_c_w;
        float J_dif_n = -ACCESS_2D(d_D_y, i, j, pitch) * grad_c_n;
        float J_dif_s = -ACCESS_2D(d_D_y, i, j - 1, pitch) * grad_c_s;

        // Alpha coefficients
        float alpha_e = ACCESS_2D(d_alpha_x, i, j, pitch);
        float alpha_w = 1.0f - ACCESS_2D(d_alpha_x, i - 1, j, pitch);
        float alpha_n = ACCESS_2D(d_alpha_y, i, j, pitch);
        float alpha_s = 1.0f - ACCESS_2D(d_alpha_y, i, j - 1, pitch);

        // Concentrations at faces
        float c_e = c_E * alpha_e + c_P * (1.0f - alpha_e);
        float c_w = c_W * alpha_w + c_P * (1.0f - alpha_w);
        float c_n = c_N * alpha_n + c_P * (1.0f - alpha_n);
        float c_s = c_S * alpha_s + c_P * (1.0f - alpha_s);

        // Advection fluxes
        float J_adv_e = -ACCESS_2D(d_D_x, i, j, pitch) * ACCESS_2D(d_dU_x, i, j, pitch) * c_e;
        float J_adv_w = -ACCESS_2D(d_D_x, i - 1, j, pitch) * ACCESS_2D(d_dU_x, i - 1, j, pitch) * c_w;
        float J_adv_n = -ACCESS_2D(d_D_y, i, j, pitch) * ACCESS_2D(d_dU_y, i, j, pitch) * c_n;
        float J_adv_s = -ACCESS_2D(d_D_y, i, j - 1, pitch) * ACCESS_2D(d_dU_y, i, j - 1, pitch) * c_s;

        // Total fluxes
        float J_E = J_dif_e + J_adv_e;
        float J_W = J_dif_w + J_adv_w;
        float J_N = J_dif_n + J_adv_n;
        float J_S = J_dif_s + J_adv_s;

        float J_tot = -J_E + J_W - d_lambda_n[j] * J_N + d_lambda_s[j] * J_S;

        // Update divergence of flux and concentration
        ACCESS_2D(d_div_J, i, j, pitch) = -J_tot;
        ACCESS_2D(d_c_next, i, j, pitch) = c_P + J_tot * d_dt;
    }
}

// Macro for allocating and copying data to device memory
// Creates pointer to an array on the device with a prefix d_
#define ALLOC2D_AND_COPY_TO_DEVICE(X)                                           \
    auto d_##X = make_unique_ptr_cuda();                         \
    hipMemcpy2D(d_##X.get(), pitch, X, Cols * sizeof(float), Cols * sizeof(float), Rows, hipMemcpyHostToDevice);

#define ALLOC1D_AND_COPY_TO_DEVICE(X)                                           \
    auto d_##X = make_unique_ptr_1D_column_vector_cuda();                         \
    hipMemcpy(d_##X.get(), X, Rows * sizeof(float), hipMemcpyHostToDevice);


int main(int argc, char* argv[]) {
    // Validate and parse command-line arguments
    if (argc != 3) {
        std::cerr << "Usage: " << argv[0] << " <number of iterations> <update_every>" << std::endl;
        return 1;
    }
    int iterations = std::atoi(argv[1]);
    int update_every = std::atoi(argv[2]);
    if (iterations <= 0 || update_every <= 0) {
        std::cerr << "Number of iterations and update_every must be positive integers." << std::endl;
        return 1;
    }

    // Copy constants to device
    hipMemcpyToSymbol(HIP_SYMBOL(d_dt), &dt, sizeof(float));
    hipMemcpyToSymbol(HIP_SYMBOL(d_timestep), &timestep, sizeof(float));

    // Allocate and copy to device memory
    ALLOC2D_AND_COPY_TO_DEVICE(c);
    ALLOC2D_AND_COPY_TO_DEVICE(c_next);
    ALLOC2D_AND_COPY_TO_DEVICE(D_x);
    ALLOC2D_AND_COPY_TO_DEVICE(D_y);
    ALLOC2D_AND_COPY_TO_DEVICE(dU_x);
    ALLOC2D_AND_COPY_TO_DEVICE(dU_y);
    ALLOC2D_AND_COPY_TO_DEVICE(alpha_x);
    ALLOC2D_AND_COPY_TO_DEVICE(alpha_y);
    ALLOC2D_AND_COPY_TO_DEVICE(div_J);

    ALLOC1D_AND_COPY_TO_DEVICE(lambda_n);
    ALLOC1D_AND_COPY_TO_DEVICE(lambda_s);

    // Start benchmarking
    auto start_time = std::chrono::high_resolution_clock::now();

        // Main computation loop
    for (int iter = 0; iter < iterations; ++iter) {

        drift_diffusion<<<gridSize, blockSize>>>(
                                        d_c.get(), d_c_next.get(), 
                                        d_D_x.get(), d_D_y.get(), 
                                        d_alpha_x.get(), d_alpha_y.get(), 
                                        d_dU_x.get(), d_dU_y.get(), 
                                        d_lambda_n.get(), d_lambda_s.get(), 
                                        d_div_J.get(), 
                                        pitch);
        apply_boundary_conditions<<<gridSize1D, blockSize1D>>>(d_c_next.get(), pitch);

        std::swap(d_c, d_c_next);
        timestep += dt;

        // Periodically update results on host
        if ((iter + 1) % update_every == 0) {
            hipDeviceSynchronize();
            hipMemcpyToSymbol(HIP_SYMBOL(d_timestep), &timestep, sizeof(float));
            hipMemcpy2D(c, Cols * sizeof(float), d_c.get(), pitch, Cols * sizeof(float), Rows, hipMemcpyDeviceToHost);
        }
    }

    hipDeviceSynchronize();
    hipMemcpyToSymbol(HIP_SYMBOL(d_timestep), &timestep, sizeof(float));
    hipMemcpy2D(c, Cols * sizeof(float), d_c.get(), pitch, Cols * sizeof(float), Rows, hipMemcpyDeviceToHost);

     // End benchmarking
    auto end_time = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> elapsed_seconds = end_time - start_time;

    std::cout << "Done, " << iterations << " iterations in "
              << elapsed_seconds.count() << " seconds." << std::endl;

    return 0;
}
